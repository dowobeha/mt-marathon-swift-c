#include "hip/hip_runtime.h"
#include <stdio.h>


__global__
void my_sample_device_code(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


__global__
void gpu_calculate_error(float *explanatory_data, float *dependent_data, float *calculated_error, float slope, float y_intercept, int number_of_data_points) {

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (i < number_of_data_points) {
//  	calculated_error[i] = y_intercept + slope*explanatory_data[i] - dependent_data[i];
	calculated_error[i] = - dependent_data[i];
  }

}

extern "C" 
void calculate_error(float *explanatory_data, float *dependent_data, float *calculated_error, float slope, float y_intercept, int number_of_data_points) {
	gpu_calculate_error<<<10,10>>>(explanatory_data, dependent_data, calculated_error, slope, y_intercept, number_of_data_points);
}



__global__
void gpu_times_equals(float *lhs, float *rhs, int number_of_data_points) {

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (i < number_of_data_points) {
  	lhs[i] *= rhs[i];
  }

}

extern "C" 
void calculate_times_equals(float *lhs, float *rhs, int number_of_data_points) {
	gpu_times_equals<<<10,10>>>(lhs, rhs, number_of_data_points);
}

/*
extern "C"
void calculate_sum(float *data, int number_of_data_points) {
	gpu_sum
}
*/


extern "C" int my_sample_host_code( int x1, int y1 )

{

  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  my_sample_device_code<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

int result;
result = -1;
hipGetDeviceCount ( &result );
printf("CUDA device count: %d\n", result);

result = -2;
	hipGetDevice(&result);
	printf("CUDA device number: %d\n", result);

	int dev = 0;
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

	return 0;
}
